
#include <hip/hip_runtime.h>
extern "C" __global__ void add(const float* A, const float* B, float* C, int numel) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    
    
    if (idx < numel) {
        C[idx] = A[idx] + B[idx];
    }

    
}